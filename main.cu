#include "hip/hip_runtime.h"
#include "GCD.cuh"

int N  = 100;	
EVP_PKEY* pPubKey_first  = NULL;
FILE*     pemFile_first    = NULL;
EVP_PKEY* pPubKey_second  = NULL;
FILE*     pemFile_second    = NULL;
BIGNUM* first_modulus = NULL;
BIGNUM* second_modulus = NULL;
BIGNUM* GCD_result = NULL;
RSA* first_rsa = NULL;
RSA* second_rsa = NULL;
unsigned int i, j;
char *nr_first = NULL;
char *nr_second = NULL;

void init_variables(){
	pPubKey_first  = EVP_PKEY_new();
	pPubKey_second  = EVP_PKEY_new();
	first_modulus = BN_new();
	second_modulus = BN_new();
	GCD_result = BN_new();
}

void free_variables(){
	BN_free(first_modulus);
	BN_free(second_modulus);
	BN_free(GCD_result);
	EVP_PKEY_free(pPubKey_first);
    EVP_PKEY_free(pPubKey_second);
    fclose(pemFile_first);
    fclose(pemFile_second);
}

/*__global__ void GCD(int n, float a, float *x, float *y){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}*/



__global__ void
GCD(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        euclid_modulo(GCD_result, first_modulus, second_modulus);
    }
}



int main(void)
{
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
	int numElements = 50000;
	size_t size = numElements * sizeof(float);
	printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 512;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    GCD<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");


	clock_t start = clock();
	for(i=1;i<=N;i++){
		for(j=(i+1);j<=N;j++){
			init_variables();
			asprintf(&nr_first, "keys_and_messages/%d.pem", i);
			asprintf(&nr_second, "keys_and_messages/%d.pem", j);


			if((pemFile_first = fopen(nr_first,"rt")) && (pPubKey_first = PEM_read_PUBKEY(pemFile_first,NULL,NULL,NULL))){
		        //fprintf(stderr,"Public key read.\n");
		    }
		    else
		    {
		        fprintf(stderr,"Cannot read \"public key\".\n");

		    }

		    if((pemFile_second = fopen(nr_second,"rt")) && (pPubKey_second = PEM_read_PUBKEY(pemFile_second,NULL,NULL,NULL))){
		        //fprintf(stderr,"Public key read.\n");
		    } else {
		        fprintf(stderr,"Cannot read \"public key\".\n");

		    }

			first_rsa = EVP_PKEY_get1_RSA(pPubKey_first);
			second_rsa = EVP_PKEY_get1_RSA(pPubKey_second);
			first_modulus = first_rsa->n;
			second_modulus = second_rsa->n;
			euclid_modulo(GCD_result, first_modulus, second_modulus);
			if(!BN_is_one(GCD_result)){
				printf("%s and %s:\n", nr_first, nr_second);
				fprintf(stdout, "GCD result:\n%s\n", BN_bn2dec(GCD_result));
			}

			free_variables();
		}
	}
	clock_t stop = clock();
	double elapsed = (double)(stop - start) * 1000.0 / CLOCKS_PER_SEC;
	printf("Time elapsed in ms: %f\n", elapsed);

	return 0;
}

