#include "hip/hip_runtime.h"
#include "cuPrintf.cu"
#include "test.h"
#include "cuda_bignum.h"
//#include <openssl/bn.h>
//The macro CUPRINTF is defined for architectures
//with different compute capabilities.


#if __CUDA_ARCH__ < 200     //Compute capability 1.x architectures
#define CUPRINTF cuPrintf
#else                       //Compute capability 2.x architectures
#define CUPRINTF(fmt, ...) printf("[%d, %d]:\t" fmt, \
                                  blockIdx.y*gridDim.x+blockIdx.x,\
                                  threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
                                  __VA_ARGS__)
#endif

#define MAIN_COMPUTATIONS 1


__device__ int cu_dev_BN_ucmp(const VQ_VECTOR *a, const VQ_VECTOR *b){

    int i;
    unsigned t1, t2, *ap, *bp;

    i = a->top - b->top;
    if (i != 0)
        return (i);
    ap = a->d;
    bp = b->d;
    for (i = a->top - 1; i >= 0; i--) {
        t1 = ap[i];
        t2 = bp[i];
        if (t1 != t2)
            return ((t1 > t2) ? 1 : -1);
    }
    return (0);

}

__device__ long cu_dev_long_abs(long number){

    if(number<0)
        return -number;
    else
        return number;

}

__device__ int cu_dev_bn_usub(const VQ_VECTOR *a, const VQ_VECTOR *b, VQ_VECTOR *r){

    unsigned max, min, dif;
    register unsigned t1, t2, *ap, *bp, *rp;
    int i, carry;

    if(NULL == a || NULL == b || NULL == r)
        return 0;

    if(NULL == a->d || NULL == b->d || NULL == r->d)
        return 0;

    max = a->top;
    min = b->top;
    dif = cu_dev_long_abs(max - min);

    ap = a->d;
    bp = b->d;
    rp = r->d;

#if 1
    carry = 0;
    for (i = min; i != 0; i--) {
        t1 = *(ap++);
        t2 = *(bp++);
        if (carry) {
            carry = (t1 <= t2);
            t1 = (t1 - t2 - 1) & BN_MASK2;
        } else {
            carry = (t1 < t2);
            t1 = (t1 - t2) & BN_MASK2;
        }
# if defined(IRIX_CC_BUG) && !defined(LINT)
        dummy = t1;
# endif
        *(rp++) = t1 & BN_MASK2;
    }
#else
    carry = bn_sub_words(rp, ap, bp, min);
    ap += min;
    bp += min;
    rp += min;
#endif
    if (carry) {     
        if (!dif)

            return 0;
        while (dif) {
            dif--;
            t1 = *(ap++);
            t2 = (t1 - 1) & BN_MASK2;
            *(rp++) = t2;
            if (t1)
                break;
        }
    }
#if 0
    memcpy(rp, ap, sizeof(*rp) * (max - i));
#else
    if (rp != ap) {
        for (;;) {
            if (!dif--)
                break;
            rp[0] = ap[0];
            if (!dif--)
                break;
            rp[1] = ap[1];
            if (!dif--)
                break;
            rp[2] = ap[2];
            if (!dif--)
                break;
            rp[3] = ap[3];
            rp += 4;
            ap += 4;
        }
    }
#endif

    r->top = max;
    cu_bn_correct_top(r);
    return (1);

}


__device__ int cu_dev_bn_rshift1(VQ_VECTOR *a){


    if(NULL == a)
        return 0;

    if(NULL == a->d)
        return 0;

    if (BN_is_zero(a))
        return 0;

    unsigned *ap, *rp , t, c;
    int i, j;

    i = a->top;
    ap = a->d;

    j = i - (ap[i - 1] == 1);

    rp = a->d;
    t = ap[--i];
    c = (t & 1) ? CU_BN_TBIT : 0;
    if (t >>= 1)
        rp[i] = t;
    while (i > 0) {
        t = ap[--i];
        rp[i] = ((t >> 1) & CU_BN_MASK2) | c;
        c = (t & 1) ? CU_BN_TBIT : 0;
    }
    a->top = j;
    return (1);

}

__device__ int cu_dev_bn_lshift(VQ_VECTOR *a, unsigned n){

    if(NULL == a)
        return 0;

    if(NULL == a->d)
        return 0;

    if (BN_is_zero(a))
        return 0;

    if (0 == n)
        return 0;

    unsigned nw = 0, lb, rb, l;
    int i;
    unsigned nwb = 0, c = 0;

    nw = (n / CU_BN_BITS2);
    lb = (n % CU_BN_BITS2);
    rb = (CU_BN_BITS2 - lb);

    l=a->d[a->top-1];
    if( (l >> rb) > 0 ) nwb = 1;
    if(nw || nwb){
        //a->d = (unsigned*)realloc(a->d, (a->top + nw + nwb)*sizeof(unsigned)) ;
        a->d[a->top]=0;
        //memset((a->d+a->top-1), 0, (nw + nwb));
        //memset(a->d, 0, (nw + nwb)*sizeof(unsigned));
    }

    if (lb == 0 && nw != 0 ){
        for (i = a->top - 1; i >= 0; i--){
            a->d[nw + i] = a->d[i];
        }
    } else {
        for (i = 0; i < (a->top + nw + nwb); i++) {
            l = a->d[i];
            a->d[i] = (l << lb) | c;
            c = (l >> rb);

        }

    }
    a->top += (nw + nwb);
    return (1);

}


__device__ VQ_VECTOR *cu_dev_euclid(VQ_VECTOR *a, VQ_VECTOR *b){
    VQ_VECTOR *t = NULL;
    unsigned shifts = 0;
    while (!CU_BN_is_zero(b)) {
        if (cu_BN_is_odd(a)) {
            if (cu_BN_is_odd(b)) {
                cu_dev_bn_usub(a, b, a);
                cu_dev_bn_rshift1(a);
                if (cu_dev_BN_ucmp(a, b) < 0) {
                    t = a;
                    a = b;
                    b = t;
                }
            } else {      
                cu_dev_bn_rshift1(b);
                if (cu_dev_BN_ucmp(a, b) < 0) {
                    t = a;
                    a = b;
                    b = t;
                }
            }
        } else {              
            if (cu_BN_is_odd(b)) {
                cu_dev_bn_rshift1(a);
                if (cu_dev_BN_ucmp(a, b) < 0) {
                    t = a;
                    a = b;
                    b = t;
                }
            } else {      
                cu_dev_bn_rshift1(a);
                cu_dev_bn_rshift1(b);
                shifts++;
            }
        }
    }

    if (shifts) {
        cu_dev_bn_lshift(a, shifts);
    }
    return (a);

}

__global__ void testKernel(VQ_VECTOR *A, VQ_VECTOR *B, VQ_VECTOR *C, int N){
    int i= blockIdx.x * blockDim.x + threadIdx.x;
    //int p;
    //for(int k=0; k<N; k++)
    VQ_VECTOR *TMP;
    //cu_dev_bn_usub(&A[i], &B[i], &C[i]);
    //cu_dev_bn_lshift(&C[i], 4);
    TMP = cu_dev_euclid(&A[i], &B[i]);
    CUPRINTF("testKernel entrance by the global threadIdx= %d value: %u\n", i , TMP->d[0]);
    CUPRINTF("testKernel entrance by the global threadIdx= %d value: %u\n", i , TMP->d[1]);
    C[0] = *TMP;

    //CUPRINTF("testKernel entrance by the global threadIdx= %d value: %u\n", i , cu_dev_euclid(&A[i], &B[i])->d[1]);
    //p = cu_dev_bn_usub(dev_A[i], dev_B[i], dev_C[i]);
    //cuPrintf("testKernel: %d\n", p);
}

int main(void){
    int L = 5, //.Data length
        N = 1;

    unit_test(); //check all host bn functions

    VQ_VECTOR   *A;
    VQ_VECTOR   *device_VQ_VECTOR_A;
    VQ_VECTOR   *B;
    VQ_VECTOR   *device_VQ_VECTOR_B;
    VQ_VECTOR   *C;
    VQ_VECTOR   *device_VQ_VECTOR_C;

    hipError_t cudaStatus;

    A =   (VQ_VECTOR*)malloc(N*sizeof(VQ_VECTOR));
    B =   (VQ_VECTOR*)malloc(N*sizeof(VQ_VECTOR));
    C =   (VQ_VECTOR*)malloc(N*sizeof(VQ_VECTOR));

    for(int i=0; i<N; i++){
        VQ_VECTOR a;
        VQ_VECTOR b;
        VQ_VECTOR c;
        a.d = (unsigned*)malloc(L*sizeof(unsigned));
        b.d = (unsigned*)malloc(L*sizeof(unsigned));
        c.d = (unsigned*)malloc(L*sizeof(unsigned));
        a.top =   L;
        b.top =   L;
        c.top =   L;

        for(int j=0; j<L; j++)
            a.d[j]=0;

        for(int j=0; j<L; j++)
            b.d[j]=0;

        for(int j=0; j<L; j++)
            c.d[j]=0;

        A[i] = a;
        B[i] = b;
        C[i] = c;
    }

    cu_BN_dec2bn(&A[0], "858238501677248042531768818944");
    cu_BN_dec2bn(&B[0], "8353015802438879251643065122143616");
    L=A[0].top;
    L=B[0].top;
    L=C[0].top;
    //Prinf of all the elements of A
    /*for(int i=0; i<N; i++){
        printf("\nA[%d]={", i);
        for(int j=0; j<L; j++)
            printf("%u ",A[i].d[j]);
        printf("}\n");
    }
    printf("\n\n");*/
    //I Allocate and Copy data from A to device_VQ_VECTORon the GPU memory

    hipDeviceReset();
    cudaStatus = hipMalloc((void**)&device_VQ_VECTOR_A, N*sizeof(VQ_VECTOR));    
    cudaStatus = hipMalloc((void**)&device_VQ_VECTOR_B, N*sizeof(VQ_VECTOR));
    cudaStatus = hipMalloc((void**)&device_VQ_VECTOR_C, N*sizeof(VQ_VECTOR));
    cudaStatus = hipMemcpy(device_VQ_VECTOR_A, A, N*sizeof(VQ_VECTOR), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(device_VQ_VECTOR_B, B, N*sizeof(VQ_VECTOR), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(device_VQ_VECTOR_C, C, N*sizeof(VQ_VECTOR), hipMemcpyHostToDevice);

    for(int i = 0; i != N; ++i) {
        unsigned long *out;
        hipMalloc(&out, L*sizeof(unsigned));
        hipMemcpy(out, A[i].d, L*sizeof(unsigned), hipMemcpyHostToDevice);
        hipMemcpy(&device_VQ_VECTOR_A[i].d, &out, sizeof(void*), hipMemcpyHostToDevice);

        hipMalloc(&out, L*sizeof(unsigned));
        hipMemcpy(out, B[i].d, L*sizeof(unsigned), hipMemcpyHostToDevice);
        hipMemcpy(&device_VQ_VECTOR_B[i].d, &out, sizeof(void*), hipMemcpyHostToDevice);

        hipMalloc(&out, L*sizeof(unsigned));
        hipMemcpy(out, C[i].d, L*sizeof(unsigned), hipMemcpyHostToDevice);
        hipMemcpy(&device_VQ_VECTOR_C[i].d, &out, sizeof(void*), hipMemcpyHostToDevice);

        // will re-allocate later, for simplicity sake
        free(A[i].d);
        free(B[i].d);
        free(C[i].d);
    }

    cudaPrintfInit();
    testKernel<<<1,N>>>(device_VQ_VECTOR_A, device_VQ_VECTOR_B, device_VQ_VECTOR_C, N);//to test and see on a sigle thread
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n testKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    cudaStatus = hipMemcpy(A, device_VQ_VECTOR_A, N*sizeof(VQ_VECTOR), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(B, device_VQ_VECTOR_B, N*sizeof(VQ_VECTOR), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(C, device_VQ_VECTOR_C, N*sizeof(VQ_VECTOR), hipMemcpyDeviceToHost);


    for(int i = 0; i != N; ++i) {
        unsigned *array = (unsigned*)malloc(L*sizeof(unsigned));
        hipMemcpy(array, A[i].d, L*sizeof(unsigned), hipMemcpyDeviceToHost);
        A[i].d = array;

        hipMemcpy(array, B[i].d, L*sizeof(unsigned), hipMemcpyDeviceToHost);
        B[i].d = array;

        hipMemcpy(array, C[i].d, L*sizeof(unsigned), hipMemcpyDeviceToHost);
        C[i].d = array;
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n testKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    printf("cuda_kernel result c[0]=%s\n", cu_bn_bn2hex(&C[0]));

    hipFree(device_VQ_VECTOR_A);
    hipFree(device_VQ_VECTOR_B);
    hipFree(device_VQ_VECTOR_C);
    return 0;
}